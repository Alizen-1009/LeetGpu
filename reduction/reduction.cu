#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <torch/types.h>

#include <algorithm>
#include <vector>

#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])

// #include "solve.h"
#define CEIL(a, b) ((a + b - 1) / (b))

template <const int WARP_SIZE = 32>
__device__ __forceinline__ float WarpReduceSum(float value) {
    for (int mask = WARP_SIZE >> 1; mask; mask >>= 1) {
        value += __shfl_xor_sync(0xffffffff, value, mask);
    }
    return value;
}

template <const int WARP_SIZE = 32>
__device__ __forceinline__ float BlockReduceSum(float value) {
    int NUM_WARPS = CEIL(blockDim.x, WARP_SIZE);
    static __shared__ float shared[32];
    value = WarpReduceSum(value);
    int laneid = threadIdx.x % WARP_SIZE;
    int warpid = threadIdx.x / WARP_SIZE;

    if (laneid == 0) shared[warpid] = value;
    __syncthreads();

    value = (threadIdx.x < NUM_WARPS) ? shared[laneid] : 0.0f;
    if (warpid == 0) value = WarpReduceSum(value);
    return value;
}
template <const int THREAD_PER_BLOCK = 256, const int WARP_SIZE = 32>
__global__ void reduce_sum_kernel(const float *d_input, float *d_output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float val = 0.0f;
    if (idx < N) val = d_input[idx];
    val = BlockReduceSum(val);
    if (threadIdx.x == 0) d_output[blockIdx.x] = val;
}
template <const int THREAD_PER_BLOCK = 256, const int WARP_SIZE = 32>
__global__ void sum_kernel(float *d_input, float *d_output, int N) {
    float val = 0.0f;
    for (int i = threadIdx.x; i < N; i += blockDim.x) {
        val += d_input[i];
    }
    val = BlockReduceSum(val);
    if (threadIdx.x == 0) *d_output = val;
}
void solve(const float *input, float *output, int N) {
    float *d_input, *d_output;
    float *d_sum_partials;

    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    hipMalloc((void **)&d_input, N * sizeof(float));
    hipMalloc((void **)&d_sum_partials, blocksPerGrid * sizeof(float));
    hipMalloc((void **)&d_output, sizeof(float));
    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    reduce_sum_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_sum_partials, N);
    sum_kernel<<<1, threadsPerBlock>>>(d_sum_partials, d_output, blocksPerGrid);

    hipMemcpy(output, d_output, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_sum_partials);
}

torch::Tensor reduce_sum(torch::Tensor a) {
    TORCH_CHECK(a.is_cuda(), "Input tensor must be on CUDA device");
    TORCH_CHECK(a.dim() == 1, "Input tensor must be 1-dimensional");

    const int64_t N = a.numel();

    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // 确保中间张量也在同一设备上
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(a.device());
    torch::Tensor d_sum_partials = torch::empty({blocksPerGrid}, options);
    torch::Tensor d_sum = torch::empty({1}, options);

    // 获取原始指针
    const float *input_ptr = a.data_ptr<float>();
    float *sum_partials_ptr = d_sum_partials.data_ptr<float>();
    float *sum_ptr = d_sum.data_ptr<float>();

    reduce_sum_kernel<<<blocksPerGrid, threadsPerBlock>>>(input_ptr, sum_partials_ptr, N);
    sum_kernel<<<1, threadsPerBlock>>>(sum_partials_ptr, sum_ptr, blocksPerGrid);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return d_sum;
}

#define STRINGFY(x) #x
#define TORCH_BINDING_COMMON_EXTENSION(func) m.def(STRINGFY(func), &func, STRINGFY(func));
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) { TORCH_BINDING_COMMON_EXTENSION(reduce_sum) }